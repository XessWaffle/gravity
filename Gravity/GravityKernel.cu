#include "hip/hip_runtime.h"
#include "GravityKernel.cuh"

#include<ctime>
#include<iostream>
#include<stdio.h>
#include<hip/hip_runtime.h>

#include "hip/hip_runtime.h"
#include ""

#define THREADS_PER_BLOCK 64

__global__ void acceleration_kernel(Particle* particle, float* acc_sum, Particle* particles, int count) {

	int positionID = blockDim.x * blockIdx.x + threadIdx.x;
	
	float gConstant = 5;

	if (positionID < count) {
		Particle ref = particles[positionID];

		float distX = ref.position[0] - particle->position[0];
		float distY = ref.position[1] - particle->position[1];

		float dist2 = distX * distX + distY * distY;
		
		float force = 0;

		if (dist2 > 100) {
			force = gConstant * ref.mass * particle->mass / (float)dist2;

			float rad = atan2(distY, distX);
		
			acc_sum[positionID] = force * cos(rad);
			acc_sum[positionID + Constant::NUM_PARTICLES] = force * sin(rad);
		}
		else {
			acc_sum[positionID] = 0;
			acc_sum[positionID + Constant::NUM_PARTICLES] = 0;
		}

	}

}

__global__ void sum_kernel(float* acc_sum) {

	const int tid = threadIdx.x;

	auto step_size = 1;
	int number_of_threads = blockDim.x;

	if (tid < Constant::NUM_PARTICLES) {

		while (number_of_threads > 0)
		{
			if (tid < number_of_threads) // still alive?
			{
				const auto fst = tid * step_size * 2;
				const auto snd = fst + step_size;
				acc_sum[fst] += acc_sum[snd];
			}

			step_size <<= 1;
			number_of_threads >>= 1;
		}
	}

}

__global__ void gravity_kernel(Particle* particles, float* acc_sum, int count) {

	float dt = .05;
	float drag = 0.01;

	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < count) {

		const int blocks = Constant::NUM_PARTICLES / THREADS_PER_BLOCK + 1;
		const int threads = Constant::NUM_PARTICLES / 2;

		acceleration_kernel << <blocks, THREADS_PER_BLOCK >> > (&particles[id], (acc_sum + 2 * id * Constant::NUM_PARTICLES), particles, count);

		sum_kernel << <1, threads >> > ((acc_sum + 2 * id * Constant::NUM_PARTICLES));
		sum_kernel << <1, threads >> > ((acc_sum + (2 * id + 1) * Constant::NUM_PARTICLES));

		float x = 0, y = 0;

		/*for (int i = 0; i < Constant::NUM_PARTICLES; i++) {
			x += acc_sum[2 * id * Constant::NUM_PARTICLES];
			y += acc_sum[2 * id * Constant::NUM_PARTICLES + Constant::NUM_PARTICLES];
		}*/

		x = acc_sum[2 * id * Constant::NUM_PARTICLES];
		y = acc_sum[(2 * id + 1) * Constant::NUM_PARTICLES];

		Particle ref = particles[id];

		float* vel = ref.velocity;
		float* pos = ref.position;

		vel[0] += x * dt;
		vel[1] += y * dt;

		vel[0] -= vel[0] * dt * drag;
		vel[1] -= vel[1] * dt * drag;

		pos[0] += vel[0] * dt;
		pos[1] += vel[1] * dt;

	}
}


GravityKernel::GravityKernel() {
	this->_world = new World();
}

GravityKernel::GravityKernel(World* world) {
	this->_world = world;
}

__host__ void GravityKernel::cudaPrep() {
	Particle* particles = this->_world->particles;

	Particle* d_particles;
	float* d_acc_sum;

	float** d_positions = new float* [Constant::NUM_PARTICLES];
	float** d_velocities = new float* [Constant::NUM_PARTICLES];
	
	if (hipMalloc(&d_particles, sizeof(Particle) * Constant::NUM_PARTICLES) != hipSuccess) {
		std::cout << "Particle Device Allocation Error" << std::endl;
		return;
	}

	if (hipMalloc(&d_acc_sum, sizeof(float) * Constant::NUM_PARTICLES * Constant::NUM_PARTICLES * Constant::DIMENSIONS) != hipSuccess) {
		std::cout << "Particle Accelerations Allocation Error" << std::endl;
		return;
	}


	for (int i = 0; i < Constant::NUM_PARTICLES; i++) {

		if (hipMalloc(&(d_positions[i]), sizeof(float) * Constant::DIMENSIONS) != hipSuccess) {
			std::cout << "Position Mapping Failure" << std::endl;
		}

		if (hipMalloc(&(d_velocities[i]), sizeof(float) * Constant::DIMENSIONS) != hipSuccess) {
			std::cout << "Velocity Mapping Failure" << std::endl;
		}


		if (hipMemcpy(&(d_particles[i].position), &(d_positions[i]), sizeof(float*), hipMemcpyHostToDevice) != hipSuccess){
			std::cout << "Particle Position Allocation Error" << std::endl;
		}

		if (hipMemcpy(&(d_particles[i].velocity), &(d_velocities[i]), sizeof(float*), hipMemcpyHostToDevice) != hipSuccess) {
			std::cout << "Particle Velocity Allocation Error" << std::endl;
		}

	}

	for (int i = 0; i < Constant::NUM_PARTICLES; i++) {
		if (hipMemcpy(d_positions[i], particles[i].position, sizeof(float) * Constant::DIMENSIONS, hipMemcpyHostToDevice) != hipSuccess) {
			std::cout << "Particle Position Allocation Error" << std::endl;
		}

		if (hipMemcpy(d_velocities[i], particles[i].velocity, sizeof(float) * Constant::DIMENSIONS, hipMemcpyHostToDevice) != hipSuccess) {
			std::cout << "Particle Velocity Allocation Error" << std::endl;
		}

		if (hipMemcpy(&(d_particles[i].mass), &(particles[i].mass), sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
			std::cout << "Particle Mass Allocation Error" << std::endl;
		}
	}
	
	this->d_particles = d_particles;
	this->d_acc_sum = d_acc_sum;
	this->d_positions = d_positions;
	this->d_velocities = d_velocities;
}

__host__ void GravityKernel::runKernel() {
	
	int blocks = Constant::NUM_PARTICLES / THREADS_PER_BLOCK + 1;

	gravity_kernel<<<blocks, THREADS_PER_BLOCK>>>(d_particles, d_acc_sum, Constant::NUM_PARTICLES);
	hipDeviceSynchronize();

	Particle* ref = this->_world->particles;

	for (int i = 0; i < Constant::NUM_PARTICLES; i++) {
		hipMemcpy(ref[i].position, d_positions[i], sizeof(float) * Constant::DIMENSIONS, hipMemcpyDeviceToHost);
		hipMemcpy(ref[i].velocity, d_velocities[i], sizeof(float) * Constant::DIMENSIONS, hipMemcpyDeviceToHost);
	}

}


__host__ void GravityKernel::cudaClear() {

	std::cout << "Clearing memory" << std::endl;

	for (int i = 0; i < Constant::NUM_PARTICLES; i++) {
		hipFree(&(d_particles[i].position));
		hipFree(&(d_particles[i].velocity));
	}

	hipFree(d_particles);
}

